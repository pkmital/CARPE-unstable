// Matlab interface and host (CPU-side) code for doing GMM estimation.
// GPU-side code is in gmm_kernel.cu.
//
// Based on Mex Code from lecture slides here:
// http://www.nr.com/CS395T/lectures2008/11-GaussianMixtureModelsAndEM.pdf
//
// Modified for GPU support by Andrew Harp (andrew.harp@gmail.com)
// http://andrewharp.com/gmmcuda

#include "mex.h"
#include "nr3matlab.h"
#include "gpugaumixmod.h"
 
GpuGaumixmod *gpugmm = NULL;

/*
 * Convert a Fortran array(W,X,Y) to a C++ array[W][X][Y]
 */
void Fortran_to_C(Mat3DSing& dest, Doub* src) {
  Int d1 = dest.dim1();
  Int d2 = dest.dim2();
  Int d3 = dest.dim3();
  for (Int i = 0; i < d1; i++) {
    for (Int j = 0; j < d2; j++) {
      for (Int k = 0; k < d3; k++) {
        dest[i][j][k] = (Sing)(*src++);
      }
    }
  }
}

/*
 * Convert a C++ array[W][X][Y] to a Fortran array(W,X,Y)
 */
void C_to_Fortran(Doub* dest, Mat3DSing& src) {
  for (Int i = 0; i < src.dim1(); i++) {
    for (Int j = 0; j < src.dim2(); j++) {
      for (Int k = 0; k < src.dim3(); k++) {
        *(dest++) = src[i][j][k];
      }
    }
  }
}

/* Matlab usage: 
  gpugmm('construct',data, means)
  
  // Copy data to GPU.
  gpugmm('togpu');
  
  // Run computation on GPU.
  loglike = gpugmm('gstep',nsteps)
  
  // Copy resulting model back for analysis/display.
  gpugmm('fromgpu');
  
  [mean sig] = gpugmm(k)
  resp = gpugmm('response')
  gpugmm('delete')
*/
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  int i, j, nn, kk, mm;
  int the_result = 0; 
  if (gpugmm) {
    nn=gpugmm->nn; 
    kk=gpugmm->kk; 
    mm=gpugmm->mm;
  
    for (int i_try = 0; i_try < gpugmm->n_tries; ++i_try) {
      if (gpugmm->loglikes[i_try] > gpugmm->loglikes[the_result]) {
        the_result = i_try;
      }
    }

    //printf("!! Found best loglike(%d): %0.2f\n",  best_result, gpugmm->loglike[best_result][0]);
  }
    
  if (gpugmm && (nrhs == 1 || nrhs == 2) && mxT(prhs[0]) == mxT<Doub>()) { // [mean sig] = gpugmm(k)
    Int k = Int( mxScalar<Doub>(prhs[0]) );
    
    // If they are requesting a result specifically, let them have it.
    if (nrhs > 1) {
      the_result = Int( mxScalar<Doub>(prhs[1]) );
    }
    
    if (nlhs > 0) {
      VecDoub mean(mm, plhs[0]);
      for (i=0;i<mm;i++) {
        mean[i] = gpugmm->means[the_result][k-1][i]; // k argument comes in 1-based
      }
    }
    
    if (nlhs > 1) {
      MatDoub sig(mm, mm, plhs[1]);
      for (i=0;i<mm;i++) {
        for (j=0;j<mm;j++) {
          sig[i][j] = gpugmm->sig[the_result][k-1][(i*mm)+j];
          //printf("%0.2f ", gpugmm->sig[best_result][k-1][(i*mm)+j]);
        }
        //printf("\n");
      }
    }
  } else if (nrhs == 1 && mxScalar<char>(prhs[0]) == 't') {        // gpugmm('togpu')
    gpugmm->copyToGpu();
  } else if (nrhs == 1 && mxScalar<char>(prhs[0]) == 'f') {        // gpugmm('fromgpu')
    gpugmm->copyFromGpu();
  } else if (nrhs == 1 && mxScalar<char>(prhs[0]) == 'd') {        // gpugmm('delete')
    if (gpugmm != NULL) {
      delete gpugmm;
    } else {
      printf("GMM was already NULL, can't delete!\n");
    }
    gpugmm = NULL;
  } else if (gpugmm && (nrhs == 1 || nrhs == 2) && mxScalar<char>(prhs[0]) == 'r') { // gpugmm('response')
    // If they are requesting a result specifically, let them have it.
    if (nrhs > 1) {
      the_result = Int( mxScalar<Doub>(prhs[1]) );
    }
    
    if (nlhs > 0) {
      MatDoub resp(nn, kk, plhs[0]);
      for (i=0;i<nn;i++) for (j=0;j<kk;j++) resp[i][j] = gpugmm->resp[the_result][i][j];
    }
  } else if (gpugmm && nrhs == 2 && mxT(prhs[1]) == mxT<Doub>() && (mxScalar<char>(prhs[0]) == 's' || mxScalar<char>(prhs[0]) == 'g')) { // deltaloglike = gpugmm('step',nsteps)
    if (DEBUGME) {
      printf("GPU method.\n");
    }
    
    Int nstep = Int(mxScalar<Doub>(prhs[1]));
    
    VecSing tmp(gpugmm->n_tries);
    for (i=0;i<nstep;i++) {
      tmp = gpugmm->estep();
      gpugmm->mstep();
    }
    
    if (nlhs > 0) {
      VecSing loglikes(gpugmm->n_tries, plhs[0]);
      for (i = 0; i < gpugmm->n_tries; ++i) {
        loglikes[i] = tmp[i];
      }
    }
  } else if (nrhs == 3 && mxT(prhs[0]) == mxT<char>() && mxScalar<char>(prhs[0]) == 'c') { // gpugmm('construct', data, means)
    if (gpugmm) {
      delete gpugmm;
    }
    
    MatSing data;
    
    int num_dims = mxGetNumberOfDimensions(prhs[2]);
    const Int *dimension_array = mxGetDimensions(prhs[2]);

    Mat3DSing means(num_dims > 2 ? dimension_array[2] : 1, 
                    dimension_array[1], 
                    dimension_array[0]);
        
    if (mxT(prhs[1]) == mxT<Doub>()) {
      if (DEBUGME) {
        printf("Constructing from doubles!\n");
      }
      
      MatDoub data_d(prhs[1]);
      data.resize(data_d.nrows(), data_d.ncols());
      for (int i = 0; i < data_d.nrows(); ++i)
        for (int j = 0; j < data_d.ncols(); ++j)
          data[i][j] = data_d[i][j];
      
      Doub* mean_d = (Doub *)mxGetData(prhs[2]);
      Fortran_to_C(means, mean_d);
      
      //means = tmp_means;
    } else {
      // TODO: make this work for singles...
      throw("Pass doubles for now!");
    }
    
    if (means.dim3() != data.ncols()) throw("wrong dims in gpugmm 1");
    if (means.dim2() >= data.nrows()) throw("wrong dims in gpugmm 2");
    // user probably didn't transpose
    
    gpugmm = new GpuGaumixmod(data, means);
  } else {
    throw("bad call to gpugmm");
  }
  return;
}