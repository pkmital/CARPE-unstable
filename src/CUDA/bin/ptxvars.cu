#include "hip/hip_runtime.h"
/*
  nvcc ptxvars.cu -g -G --host-compilation=c -c -D__DEVICE_LAUCH_PARAMETERS_H__ -Xptxas -fext
 */
 
typedef struct dim3_16 {
    unsigned short x, y, z;
} dim3_16;

typedef struct dim2_16 {
    unsigned short x, y;
} dim2_16;

__shared__ dim3_16  threadIdx;
__shared__ dim2_16  blockIdx;
__shared__ dim3_16  blockDim;
__shared__ dim2_16  gridDim;

int $gpu_registers[128];
